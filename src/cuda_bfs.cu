#include "hip/hip_runtime.h"
#include "../include/cuda_bfs.cuh"

using namespace std;

// CUDA BFS算法中的线程数目
#define N_THREADS_PER_BLOCK (1 << 5)

// 初始化 cuda 中的数组
__global__
void init_cuda_array(int n, int *d_arr, int value, int start_index) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // 根据线程 ID 进行数组初始化
    if (tid == start_index) {
        // 起始点距离设置为0
        d_arr[start_index] = 0;
    } else if (tid < n) {
        // 其他点的距离初始化为给定的值
        d_arr[tid] = value;
    }
}

// 已知图和当前队列，寻找下一层遍历的队列
__global__
void find_next_queue(int *adjacency_list, int *edge_offset, int *edges_size, int *distance,
        int queue_size, int *cur_queue, int *next_queue_size, int *next_queue, int level) {
    
    // 获取线程信息
    const int tid = blockIdx.x * blockDim.x + threadIdx.x; 
    // 并行遍历队列
    if (tid < queue_size) {
        int current = cur_queue[tid];
        // 遍历当前节点的邻接节点
        for (int i = edge_offset[current]; i < edge_offset[current] + edges_size[current]; ++i) {
            int v = adjacency_list[i];
            if (distance[v] == INT_MAX) {
                // 如果邻接节点的距离为无穷大，更新距离并将其加入下一层队列
                distance[v] = level + 1;
                int position = atomicAdd(next_queue_size, 1);
                next_queue[position] = v;
            }
        }
    }
}

// CUDA BFS算法主函数
void cuda_bfs(int start, Graph &my_graph, vector<int> &distance, vector<bool> &is_visited) {

    const int N_BLOCKS = (my_graph.vertex_num + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK;
    const int NEXT_QUEUE_SIZE = 0;

    // 初始化 cuda 中的变量
    int *device_adjacency_list;
    int *device_edges_offset;
    int *device_edges_size;
    int *device_first_queue;
    int *device_second_queue;
    int *device_next_queue_size;
    int *device_distance_array; // output

    // 初始化位于 cpu 中的变量
    int cur_queue_size = 1;
    int level = 0;

    // 在 gpu 上分配内存空间，并输入图数据
    int vertex_size = my_graph.vertex_num * sizeof(int);
    int adjacency_list_size = my_graph.adjacency_list.size() * sizeof(int);
    hipMalloc((void **)&device_adjacency_list, adjacency_list_size);
    hipMalloc((void **)&device_edges_offset, vertex_size);
    hipMalloc((void **)&device_edges_size, vertex_size);
    hipMalloc((void **)&device_first_queue, vertex_size);
    hipMalloc((void **)&device_second_queue, vertex_size);
    hipMalloc((void **)&device_distance_array, vertex_size);
    hipMalloc((void **)&device_next_queue_size, sizeof(int));

    // 将图数据复制到 GPU 内存
    hipMemcpy(device_adjacency_list, &my_graph.adjacency_list[0], adjacency_list_size, hipMemcpyHostToDevice);
    hipMemcpy(device_edges_offset, &my_graph.edge_offset[0], vertex_size, hipMemcpyHostToDevice);
    hipMemcpy(device_edges_size, &my_graph.edges_size[0], vertex_size, hipMemcpyHostToDevice);
    hipMemcpy(device_next_queue_size, &NEXT_QUEUE_SIZE, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_first_queue, &start, sizeof(int), hipMemcpyHostToDevice);

    // 完成内存拷贝，开始运行 CUDA BFS
    clock_t start_time = clock();
    distance = vector<int> (my_graph.vertex_num, INT_MAX);
    distance[start] = 0;
    hipMemcpy(device_distance_array, distance.data(), vertex_size, hipMemcpyHostToDevice);

    while (cur_queue_size > 0) {
        int *device_cur_queue;
        int *device_next_queue;
        // 根据当前遍历的层数，选择使用哪一个队列
        if (level % 2 == 0) {
            device_cur_queue = device_first_queue;
            device_next_queue = device_second_queue;
        } else {
            device_cur_queue = device_second_queue;
            device_next_queue = device_first_queue;
        }
        // 调用 CUDA 函数，寻找下一层遍历的队列
        find_next_queue<<<N_BLOCKS, N_THREADS_PER_BLOCK>>> (device_adjacency_list, device_edges_offset, device_edges_size, device_distance_array,
                cur_queue_size, device_cur_queue, device_next_queue_size, device_next_queue, level);
        hipDeviceSynchronize();
        hipMemcpy(&cur_queue_size, device_next_queue_size, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(device_next_queue_size, &NEXT_QUEUE_SIZE, sizeof(int), hipMemcpyHostToDevice);
        level++;
    }

    // 将 GPU 计算结果复制回 CPU 内存
    hipMemcpy(&distance[0], device_distance_array, vertex_size, hipMemcpyDeviceToHost);
    clock_t end_time = clock();
    double duration = ((double)(end_time - start_time));
    printf("Elapsed time for naive linear GPU implementation (without copying graph): %.1lf ms.\n", duration);

    // 释放 GPU 内存空间
    hipFree(device_adjacency_list);
    hipFree(device_edges_offset);
    hipFree(device_edges_size);
    hipFree(device_first_queue);
